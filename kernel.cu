//verion 1
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <fstream>

#include <hip/device_functions.h>

using namespace std;
#define SIZE 10240
#define MAXVN 10


__global__ void addKernel(double *c, const double *a, const double *b)
{
    int i = threadIdx.x;
	c[i] = exp(a[i]+b[i]);
}
 
__global__ void simulatedAnnealingKernel(int *route,
										 double * costs,
										 const int solutionLen,
										 const int *demands,
										 const double * distances,
										 const int nodeNum,
										 const int capacities,
										 const int vNum,
										 const int *randInt,
										 const double *randDouble,
										 const double tempParam,
										 const double cr)
{ 
	int depot = threadIdx.x;
	int *bestSolution = route + (depot)*solutionLen;
	__shared__ int solutionArray[SIZE];

	//ÿ���̷߳���3*solutonLen*4 Byte ��С�Ĺ����ڴ�
	 
	//�����ڴ����  
	int *solution = &solutionArray[threadIdx.x*(solutionLen*2+7*MAXVN)]; //solutionLen

    int *curSolution = (int *)&solution[solutionLen];  //solutionLen
	double *dice =(double*)&curSolution[solutionLen];
	double *dis = (double*)&dice[1];
	double *newDis = (double*)&dis[1];
	int *cusIndex = (int*)&newDis[1];
	int *range = (int*)&cusIndex[1];
	int *v = (int*)&range[1];
	int *improvedTryCounter = (int*)&v[1];
    int *demandCounter=(int*)&improvedTryCounter[MAXVN]; //4
	int *routeStart=(int*)&demandCounter[MAXVN];  //6
	double *acc = (double*)&routeStart[MAXVN+1];    //5
	int *r1=v;
	int *r2=range;
	int *inter =cusIndex;

	//�Ĵ�������
    int maxDemandIndex = 0;
	double demandSum = 0;
    double minCost = 100000;
    int insertPoint = 0;
    int cus=0;
	double cost=0;
	int pre=0;
	int cur=0;
	int i=0;
	int maxDemand=0;
	int nextRand = 0;
	int strategy = 0;
	double temp = tempParam;//cr = 0.001;//tempPara;
	


	//�������
    *dis = 0;
	int len = 1;
	for(int i =1; bestSolution[i]!=0; i++){
		  pre = bestSolution[i-1];
		  cur = bestSolution[i];
		  *dis += distances[(pre-1)*nodeNum+cur-1];
		  len ++;
	}

	for(int i =0; i< len; i++){
		solution[i] = bestSolution[i];
	    curSolution[i] = bestSolution[i];
     }
	//����Ƿ���
	maxDemandIndex = 0;
	demandSum = 0;
	*v = 0;
	demandCounter[*v] = 0;
	routeStart[*v] = 0;
	for(i = 1;i<len; i++){
	   if (solution[i] == depot+1){
           if( demandCounter[*v] > demandCounter[maxDemandIndex])
                 maxDemandIndex = *v;
			(*v)++;
			demandCounter[*v] = 0;
			routeStart[*v] = i;
			}
		else{
			demandCounter[*v]+= demands[solution[i]-1];
			demandSum += demands[solution[i]-1];
		}
	}
	//�ж��½��Ƿ���������Լ��
		for(i=0; i<vNum; i++){
		   if(demandCounter[i] > capacities){
		      *dis += capacities;
		   }
		}


	costs[depot] = *dis;
	
	improvedTryCounter[0] =1;
	improvedTryCounter[1]=1;
	
	while(temp > 0.01){
	     //ѡ�����
	    *dice =randDouble[(nextRand+threadIdx.x*20)%1000];
		nextRand =(nextRand+1)%1000;
		if(*dice <= 0.1+0.8*(improvedTryCounter[0]/(double)(improvedTryCounter[0]+improvedTryCounter[1]))){
			//ѡ�����1
			strategy = 0;
		    *r1 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
		    *r2 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
			//swap *r1,*r2 in solution
		    *inter = solution[*r1];
			solution[*r1] = solution[*r2];
			solution[*r2] = *inter;
		}
		else{
		    //ѡ�����2
			//ͳ�Ƹ��������ĸ��أ����ݳ����ĸ��ص���
			strategy = 1;
			
            maxDemandIndex = 0;
			demandSum = 0;
		    *v = 0;
			demandCounter[*v] = 0;
			routeStart[*v] = 0;
			for(i = 1;i<len; i++){
			    if (solution[i] == depot+1){
                    if( demandCounter[*v] > demandCounter[maxDemandIndex])
                         maxDemandIndex = *v;
					(*v)++;
					demandCounter[*v] = 0;
					routeStart[*v] = i;
				}
				else{
					demandCounter[*v]+= demands[solution[i]-1];
				    demandSum += demands[solution[i]-1];
				}
			}
			
			acc[0] = demandCounter[0]/(demandSum+0.1);
			for(i = 1; i< vNum; i++){
			     acc[i] = acc[i-1]+demandCounter[i]/(demandSum+0.1);
			}
			acc[i-1] = 1;

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			*v = 0;
			while(*dice>acc[*v]) (*v)++; //�ҵ���Ӧ�ĳ�Ϊv
			//��v�г�ȡһ���ͻ�Ȼ������뵽�����ĳ����ʵ�·��λ�� ���ͻ���λ��Ӧ����routeStart[*v]��routeStart[v+1]֮��
		    *range = routeStart[*v+1]-routeStart[*v]-1;
			if(*range ==0)
				continue;
		    *cusIndex = routeStart[*v]+1+randInt[(nextRand+threadIdx.x*20)%1000]%*range;
			nextRand = (++nextRand)%1000;


            //������ѡ���ؽ�С�ĳ�
		    maxDemand = demandCounter[maxDemandIndex];
			acc[0] = (maxDemand - demandCounter[0])/(vNum*maxDemand - demandSum+0.1);
			for(i =1; i< vNum; i++){
			    acc[i] = acc[i-1]+(maxDemand - demandCounter[i])/(vNum*maxDemand - demandSum+0.1);
			}
			acc[i-1] = 1;

			//����demandCounter
			demandCounter[*v] -= demands[solution[*cusIndex]-1];

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;

			*v = 0;
			while(*dice>acc[*v]) (*v)++; //�ҵ���Ӧ�ĳ�Ϊ*v �䷶ΧΪrouteStart[*v]��routeStart[*v+1]

			//����demandCounter
			demandCounter[*v] += demands[solution[*cusIndex]-1];

		    minCost = 100000;
		    insertPoint = 0;
		    cus = solution[*cusIndex];
			cost = 0;
			//��*cusIndex��Ӧ�Ŀͻ����뵽����*v��Ӧ��·���к��ʵ�λ��
            for(i = routeStart[*v]; i<routeStart[*v+1]; i++){
				   cost = distances[(cus-1)*nodeNum+solution[i]-1]+
					      distances[(cus-1)*nodeNum+solution[i+1]-1]-
						  distances[(solution[i]-1)*nodeNum+solution[i+1]-1];
				   if(cost < minCost){
					   minCost =cost;
					   insertPoint = i;
				   }
			}

		    //��cus��*cusIndex��λ�ò��뵽insertPoint��λ��
			if( *cusIndex <insertPoint){
			   for(int i = *cusIndex; i <insertPoint; i++){
			         solution[i] = solution[i+1];
			   }
			   solution[insertPoint] = cus;
			}else{
			   for(int i = *cusIndex; i >insertPoint+1; i--){
			          solution[i] = solution[i-1];
			   }
			   solution[insertPoint+1] = cus;
			}
		}

		//�����½���ܾ���
	   *newDis = 0;
	    for(i =1; i<len-1;i++){
		 pre = solution[i];
		 cur = solution[i+1];
		*newDis += distances[(pre-1)*nodeNum+cur-1];
	    }
		//�ж��½��Ƿ���������Լ��
		for(i=0; i<vNum; i++){
		   if(demandCounter[i] > capacities){
		      *newDis += capacities;
		   }
		}

		//����½�ȵ�ǰ����ţ��滻
		if(*newDis < *dis){
			for(i = 0;i< len; i++){
				curSolution[i] = solution[i];
			}
			*dis = *newDis;
			improvedTryCounter[strategy] ++;
			//��������Ž���ţ��滻���Ž�
			if( *newDis < costs[depot]){
				for(i =0; i< len; i++){
				   bestSolution[i] = solution[i];
				}
				costs[depot] = *newDis;
			}
		}else{
			//�����Ը��� exp((dis - newDis)/temp)�滻
			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			if(*dice < exp((*dis - *newDis)/temp)){
			     for(i = 0;i< len; i++){
				   curSolution[i] = solution[i];
			     }
			     *dis = *newDis;
			}else{
				//����������½⣬��ԭ��
			    for(i = 0;i< len; i++){
				   solution[i] = curSolution[i];
			    }
			}
		}
		temp *=1 - cr;
		__syncthreads(); 
	}
}


int main(){
	const int nodeNum = 52;
	const int depotNum = 2;
	const int vehicleNum = 6;
	const int solutionLen = nodeNum - depotNum + vehicleNum + 1;
	int r[depotNum*solutionLen] = {0};
	int demand[nodeNum] = {0};
	double dis[nodeNum*nodeNum]={0};
	double randDouble[1000] = {0};
	int randInt[1000] ={0};
	int capacities = 5000;
	int vNum =6;


	ifstream routeStream("routes.txt");
	ifstream distanceStream("distances.txt");
	ifstream demandStream("demands.txt");
	ifstream rdStream("randDouble.txt");
	ifstream riStream("randInt.txt");
	

	for(int i=0; i<depotNum; i++){
		for(int j =0; j< solutionLen; j++){
			 routeStream>>r[i*solutionLen+j];
		}
	}
	for(int i=0; i<nodeNum; i++){
		demandStream>>demand[i];
		for(int j = 0; j< nodeNum; j++){
		    distanceStream>>dis[i*nodeNum + j];
		}
	}

	for(int i=0; i<1000; i++){
	    rdStream>>randDouble[i];
		riStream>>randInt[i];
	}
	routeStream.close();
	distanceStream.close();
	demandStream.close();
	riStream.close();
	rdStream.close();

	double costs[depotNum]={0};
	hipError_t e;

	int* d_r;
	e = hipMalloc((void**)&d_r,sizeof(int)*depotNum*solutionLen);
    e = hipMemcpy(d_r,r,sizeof(int)*depotNum*solutionLen,hipMemcpyHostToDevice);

	double* d_costs;
	e = hipMalloc((void**)&d_costs,sizeof(double)*depotNum);

	int *d_demand;
	e = hipMalloc((void**)&d_demand,sizeof(int)*nodeNum);
	e = hipMemcpy(d_demand,demand,sizeof(int)*nodeNum,hipMemcpyHostToDevice);

    double *d_dis;
	e = hipMalloc((void**)&d_dis,sizeof(double)*nodeNum*nodeNum);
	e = hipMemcpy(d_dis,dis,sizeof(double)*nodeNum*nodeNum,hipMemcpyHostToDevice);

    int *d_randInt;
    e = hipMalloc((void**)&d_randInt,sizeof(int)*1000);
	e = hipMemcpy(d_randInt,randInt,sizeof(int)*1000,hipMemcpyHostToDevice);

	double *d_randDouble;
	e = hipMalloc((void**)&d_randDouble,sizeof(double)*1000);
	e = hipMemcpy(d_randDouble,randDouble,sizeof(double)*1000,hipMemcpyHostToDevice);
	
	simulatedAnnealingKernel<<<1,depotNum>>>(d_r,d_costs,solutionLen,d_demand,d_dis,nodeNum,capacities,vNum,d_randInt,d_randDouble,100000,0.001);
	
	e = hipMemcpy(r,d_r,sizeof(int)*depotNum*solutionLen,hipMemcpyDeviceToHost);
	e = hipMemcpy(costs,d_costs,sizeof(double)*depotNum,hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_costs);
	hipFree(d_demand);
	hipFree(d_dis);
	hipFree(d_randDouble);
	hipFree(d_randInt);
  
 return 0;
}



 