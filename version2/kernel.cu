//verison 2
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <fstream>

#include <hip/device_functions.h>

using namespace std;
#define SIZE 10240
#define MAXVN 10


__global__ void addKernel(double *c, const double *a, const double *b)
{
    int i = threadIdx.x;
	c[i] = exp(a[i]+b[i]);
}
 
__global__ void simulatedAnnealingKernel(int *route,
										 double * costs,
										 const int solutionLen,
										 const int *demands,
										 const double * distances,
										 const int nodeNum,
										 const int *capacities,
										 const int *randInt,
										 const double *randDouble,
										 const double tempParam,
										 const double cr)
{ 
	int depot = threadIdx.x;
	int *bestSolution = route + (depot)*solutionLen;
	__shared__ int solutionArray[SIZE];

	//每个线程分配3*solutonLen*4 Byte 大小的共享内存
	 
	//共享内存变量  
	int *solution = &solutionArray[threadIdx.x*(solutionLen*2+6*MAXVN)]; //solutionLen
	
    int *curSolution = (int *)&solution[solutionLen];  //solutionLen
	double *dice =(double*)&curSolution[solutionLen];
	int *cusIndex = (int*)&dice[1];
	int *range = (int*)&cusIndex[1];
	int *v = (int*)&range[1];
	int *improvedTryCounter = (int*)&v[1];
    int *demandCounter=(int*)&improvedTryCounter[MAXVN]; //4
	int *routeStart=(int*)&demandCounter[MAXVN];  //6
	double *acc = (double*)&routeStart[MAXVN+1];    //5
	

	//寄存器变量
	/*int demandCounter[4]={0};
	int routeStart[6] = {0};
	double acc[5] = {0};*/

    int maxDemandIndex = 0;
	double demandSum = 0;
    double minCost = 100000;
    int insertPoint = 0;
    int cus=0;
	double cost=0;
	int pre=0;
	int cur=0;
	int i=0;
	int maxDemand=0;
	int *r1=v;
	int *r2=range;
	int *inter =cusIndex;
	int nextRand = 0;
	int strategy = 0;
	double temp = tempParam;//cr = 0.001;//tempPara;
	


	//计算距离
	double dis = 0;
	int len = 1;
	for(int i =1; bestSolution[i]!=0; i++){
		  pre = bestSolution[i-1];
		  cur = bestSolution[i];
		  dis += distances[(pre-1)*nodeNum+cur-1];
		  len ++;
	}

	for(int i =0; i< len; i++){
		solution[i] = bestSolution[i];
	    curSolution[i] = bestSolution[i];
     }

	costs[depot] = dis;
	
	improvedTryCounter[0] = 1;
	improvedTryCounter[1] = 1;
	improvedTryCounter[2] = 1;
	
	while(temp > 0.01){
	     //选择策略
	    *dice =randDouble[(nextRand+threadIdx.x*20)%1000];
		nextRand =(nextRand+1)%1000;
		if(*dice <= 1){//0.1+0.7*(improvedTryCounter[0]/(double)(improvedTryCounter[0]+improvedTryCounter[1]+improvedTryCounter[2]))
			//选择策略0
			strategy = 0;
		    *r1 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
		    *r2 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
			//swap *r1,*r2 in solution
		    *inter = solution[*r1];
			solution[*r1] = solution[*r2];
			solution[*r2] = *inter;
		}
		else if(*dice <= 0.1+0.7*((improvedTryCounter[0]+improvedTryCounter[1])/(double)(improvedTryCounter[0]+improvedTryCounter[1]+improvedTryCounter[2]))){
		    //选择策略1
			//统计各个车俩的负载，根据车辆的负载调整
			strategy = 1;
			
            maxDemandIndex = 0;
			demandSum = 0;
		    *v = 0;
			demandCounter[*v] = 0;
			routeStart[*v] = 0;
			for(i = 1;i<len; i++){
			    if (solution[i] == depot+1){
                    if( demandCounter[*v] > demandCounter[maxDemandIndex])
                         maxDemandIndex = *v;
					(*v)++;
					demandCounter[*v] = 0;
					routeStart[*v] = i;
				}
				else{
					demandCounter[*v]+= demands[solution[i]-1];
				    demandSum += demands[solution[i]-1];
				}
			}
			
			acc[0] = demandCounter[0]/(demandSum+0.1);
			for(i = 1; i< 4; i++){
			     acc[i] = acc[i-1]+demandCounter[i]/(demandSum+0.1);
			}
			acc[i-1] = 1;

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			*v = 0;
			while(*dice>acc[*v]) (*v)++; //找到对应的车为v
			//从v中抽取一个客户然后将其插入到其他的车的适当路线位置 而客户的位置应该在routeStart[*v]和routeStart[v+1]之间
		    *range = routeStart[*v+1]-routeStart[*v]-1;
			if(*range ==0)
				continue;
		    *cusIndex = routeStart[*v]+1+randInt[(nextRand+threadIdx.x*20)%1000]%*range;
			nextRand = (++nextRand)%1000;


            //按概率选择负载较小的车
		    maxDemand = demandCounter[maxDemandIndex];
			acc[0] = (maxDemand - demandCounter[0])/(4*maxDemand - demandSum+0.1);
			for(i =1; i< 4; i++){
			    acc[i] = acc[i-1]+(maxDemand - demandCounter[i])/(4*maxDemand - demandSum+0.1);
			}
			acc[i-1] = 1;

			//更新demandCounter
			demandCounter[*v] -= demands[solution[*cusIndex]-1];

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;

			*v = 0;
			while(*dice>acc[*v]) (*v)++; //找到对应的车为*v 其范围为routeStart[*v]到routeStart[*v+1]

			//更新demandCounter
			demandCounter[*v] += demands[solution[*cusIndex]-1];

		    minCost = 100000;
		    insertPoint = 0;
		    cus = solution[*cusIndex];
			cost = 0;
			//将*cusIndex对应的客户插入到车辆*v对应的路径中合适的位置
            for(i = routeStart[*v]; i<routeStart[*v+1]; i++){
				   cost = distances[(cus-1)*nodeNum+solution[i]-1]+
					      distances[(cus-1)*nodeNum+solution[i+1]-1]-
						  distances[(solution[i]-1)*nodeNum+solution[i+1]-1];
				   if(cost < minCost){
					   minCost =cost;
					   insertPoint = i;
				   }
			}

		    //将cus从*cusIndex的位置插入到insertPoint的位置
			if( *cusIndex <insertPoint){
			   for(int i = *cusIndex; i <insertPoint; i++){
			         solution[i] = solution[i+1];
			   }
			   solution[insertPoint] = cus;
			}else{
			   for(int i = *cusIndex; i >insertPoint+1; i--){
			          solution[i] = solution[i-1];
			   }
			   solution[insertPoint+1] = cus;
			}
		}
		//strategy 3 insertion
		else{
		    strategy = 2;
			*r1 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
		    *r2 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
			*inter = solution[*r1];
			if(*r1<*r2){
			  for(int i =*r1;i<*r2;i++){
			     solution[i] = solution[i+1];
			  }
			  
			}else if(*r1>*r2){
				for(int i =*r1; i>*r2; i--){
				   solution[i] = solution[i-1];
				}
			}
			solution[*r2] = *inter;
		}
		//计算新解的总距离
		double newDis = 0;
	    for(i =1; i<len-1;i++){
		  pre = solution[i];
		  cur = solution[i+1];
		 newDis += distances[(pre-1)*nodeNum+cur-1];
	    }
		//判断新解是否满足容量约束
		for(i=0; i<4; i++){
		   if(demandCounter[i] > capacities[depot]/4){
		      newDis += capacities[depot];
		   }
		}

		//如果新解比当前解更优，替换
		if(newDis < dis){
			for(i = 0;i< len; i++){
				curSolution[i] = solution[i];
			}
			dis = newDis;
			improvedTryCounter[strategy] ++;
			//如果比最优解更优，替换最优解
			if( newDis < costs[depot]){
				for(i =0; i< len; i++){
				   bestSolution[i] = solution[i];
				}
				costs[depot] = newDis;
			}
		}else{
			//否则以概率 exp((dis - newDis)/temp)替换
			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			if(*dice < exp((dis - newDis)/temp)){
			     for(i = 0;i< len; i++){
				   curSolution[i] = solution[i];
			     }
			     dis = newDis;
			}
		}
		temp *=1 - cr;
		__syncthreads(); 
	}
}


int main(){
	const int nodeNum = 97;
	const int depotNum = 9;
	const int vehicleNum = 3;
	const int solutionLen = nodeNum - depotNum + vehicleNum + 1;
	int r[depotNum*solutionLen] = {0};
	int demand[nodeNum] = {0};
	double dis[nodeNum*nodeNum]={0};
	double randDouble[1000] = {};
	int randInt[1000] ={};

	ifstream routeStream("routes.txt");
	ifstream distanceStream("distances.txt");
	ifstream demandStream("demands.txt");
	ifstream rdStream("randDouble.txt");
	ifstream riStream("randInt.txt");
	for(int i=0; i<depotNum; i++){
		for(int j =0; j< solutionLen; j++){
			 routeStream>>r[i*solutionLen+j];
		}
	}
	for(int i=0; i<nodeNum; i++){
		demandStream>>demand[i];
		for(int j = 0; j< nodeNum; j++){
		    distanceStream>>dis[i*nodeNum + j];
		}
	}

	for(int i=0; i<1000; i++){
	    rdStream>>randDouble[i];
		riStream>>randInt[i];
	}
	routeStream.close();
	distanceStream.close();
	demandStream.close();
	riStream.close();
	rdStream.close();

	double costs[depotNum]={0};
	hipError_t e;

	int* d_r;
	e = hipMalloc((void**)&d_r,sizeof(int)*depotNum*solutionLen);
    e = hipMemcpy(d_r,r,sizeof(int)*depotNum*solutionLen,hipMemcpyHostToDevice);

	double* d_costs;
	e = hipMalloc((void**)&d_costs,sizeof(double)*depotNum);

	int *d_demand;
	e = hipMalloc((void**)&d_demand,sizeof(int)*nodeNum);
	e = hipMemcpy(d_demand,demand,sizeof(int)*nodeNum,hipMemcpyHostToDevice);

    double *d_dis;
	e = hipMalloc((void**)&d_dis,sizeof(double)*nodeNum*nodeNum);
	e = hipMemcpy(d_dis,dis,sizeof(double)*nodeNum*nodeNum,hipMemcpyHostToDevice);

    int *d_randInt;
    e = hipMalloc((void**)&d_randInt,sizeof(int)*1000);
	e = hipMemcpy(d_randInt,randInt,sizeof(int)*1000,hipMemcpyHostToDevice);

	double *d_randDouble;
	e = hipMalloc((void**)&d_randDouble,sizeof(double)*1000);
	e = hipMemcpy(d_randDouble,randDouble,sizeof(double)*1000,hipMemcpyHostToDevice);
	int capacities[4] = {30000,30000,30000,30000};
	int *d_capacities;
	e = hipMalloc((void**)&d_capacities,sizeof(int)*depotNum);
	e = hipMemcpy(d_capacities,capacities,sizeof(int)*depotNum,hipMemcpyHostToDevice);

	simulatedAnnealingKernel<<<1,depotNum>>>(d_r,d_costs,solutionLen,d_demand,d_dis,nodeNum,d_capacities,d_randInt,d_randDouble,100000,0.001);
	
	e = hipMemcpy(r,d_r,sizeof(int)*depotNum*solutionLen,hipMemcpyDeviceToHost);
	e = hipMemcpy(costs,d_costs,sizeof(double)*depotNum,hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_costs);
	hipFree(d_demand);
	hipFree(d_dis);
	hipFree(d_capacities);
	hipFree(d_randDouble);
	hipFree(d_randInt);
  
 return 0;
}



 